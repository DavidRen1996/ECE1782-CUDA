
#include <hip/hip_runtime.h>
#include "sys/time.h"

#include <stdio.h>

double getTimeStamp() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

// device-side matrix addition
/*__global__ void f_addmat(float* A, float* B, float* C, int nx, int ny) {
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	int ix = threadId.x + bloackId.x * blockDim.x;
	int iy = threadId.y + bloackId.y * blockDim.y;
	int idx = iy * ny + ix;
	if ((ix < nx) && (iy < ny))
		C[idx] = A[idx] + B[idx];
}*/


__global__ void f_addmat(float* A, float* B, float* C, int nx, int ny) {
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	//int numElem=nx*ny;
	long long idx = iy * nx + ix;
	if ((ix < nx) && (iy < ny)){

		C[idx] = A[idx] + B[idx];

	}
}

void matrixSumHost(float* A, float* B, float* C, int nx, int ny)
{
	float* ia = A, * ib = B, * ic = C;
	for (int iy = 0; iy < ny; iy++) {
		for (int ix = 0; ix < nx; ix++)
			ic[ix] = ia[ix] + ib[ix];
		ia += nx; ib += nx; ic += nx;
	}
}

void initDataA(float *h_A, int nx, int ny) {
	int xcoord = 0;
	int ycoord = 0;
	for (int i = 0; i < ny*nx; i++) {
		h_A[i]= (float)(xcoord + ycoord) / 3.0;
		if (xcoord == nx - 1) {
			xcoord = 0;
			ycoord += 1;
		}
		else {
			xcoord += 1;
		}
		
	}
}

void initDataB(float* h_B, int nx, int ny) {
	int xcoord = 0;
	int ycoord = 0;
	for (int i = 0; i < ny * nx; i++) {
		h_B[i] = (float)(xcoord + ycoord)* 3.14;
		if (xcoord == nx - 1) {
			xcoord = 0;
			ycoord += 1;
		}
		else {
			xcoord += 1;
		}

	}
}
//int argc, char* argv[]
int main(int argc, char* argv[]) {

	if (argc != 3) {
		printf("Error: wrong number of args\n");
		//exit();
	}
	int nx = atoi(argv[1]); // should check validity
	int ny = atoi(argv[2]); // should check validity
	int noElems = nx * ny;
	int bytes = noElems * sizeof(float);
	//printf("my name");
	float* h_A = (float*)malloc(bytes);
	float* h_B = (float*)malloc(bytes);
	float* h_hC = (float*)malloc(bytes); // host result
	initDataA(h_A, nx,ny);
	initDataB(h_B, nx, ny);
	matrixSumHost(h_A, h_B, h_hC, nx, ny);
/* device side*/
	float* d_A, * d_B, * d_C;
	hipMalloc((void**)& d_A, bytes);
	hipMalloc((void**)& d_B, bytes);
	hipMalloc((void**)& d_C, bytes);
	double timeStampA = getTimeStamp();

	float* h_dC = (float*)malloc(bytes);

	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
	double timeStampB = getTimeStamp();
	
	int blockx=32;
	int blocky=32;
	int marker=0;
	while (nx>blockx*65535){
		marker=1;
		blockx=2*blockx;
	}
	while (ny>blocky*65535){
		marker=2;
		blocky=2*blocky;
	}
	if (marker==1){
		blocky=1024/blockx;
	}
	dim3 block(blockx, blocky); // you will want to configure this

	int gridSizeX=(nx + block.x - 1) / block.x;
	if (gridSizeX>=65535){

		gridSizeX=65535;

	}

	int gridSizeY=(ny + block.y - 1) / block.y;
	if (gridSizeY>=65535){

		gridSizeY=65535;

	}
	dim3 grid(gridSizeX, gridSizeY);

	f_addmat <<<grid, block >>> (d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();
	double timeStampC = getTimeStamp();

	hipMemcpy(h_dC, d_C, bytes, hipMemcpyDeviceToHost);

	double timeStampD = getTimeStamp();
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipDeviceReset();
	/*for (int j=0;j<10;j++){
		printf("%f %f\n",h_dC[j],h_hC[j]);
	}*/
	
	for (int i = 0; i < noElems; i++) {
		if (h_dC[i]!=h_hC[i]){

			printf("wrong %d\n",i);
			for (int j=i;j<i+10;j++){
				printf("%f %f\n",h_dC[j],h_hC[j]);
			}
			break;
		}
		
	}
	printf("total_time:%4f",timeStampD-timeStampA);
	printf("CPU_GPU_time:%4f",timeStampB-timeStampA);
	printf("kernel_time:%4f",timeStampC-timeStampB);
	printf("GPU_CPU_time:%4f",timeStampD-timeStampC);
	






//...............................................................................

	/*for (int i = 0; i < noElems; i++) {
		cout << h_A[i];
	}
	cout << " " << endl;
	for (int i = 0; i < noElems; i++) {
		cout << h_B[i];
	}
	cout << " " << endl;
	
	matrixSumHost(h_A, h_B, h_hC, nx, ny);
	for (int i = 0; i < noElems; i++) {
		cout << h_hC[i];
	}
	cout << " " << endl;*/

	//cout << h_A[900]<<endl;
/*
// get program arguments
	if (argc != 3) {
		printf("Error: wrong number of args\n");
		//exit();
	}
	int nx = atoi(argv[2]); // should check validity
	int ny = atoi(argv[3]); // should check validity
	int noElems = nx * ny;
	int bytes = noElems * sizeof(float);
	// but you may want to pad the matrices��
	// alloc memory host-side
	float* h_A = (float*)malloc(bytes);
	float* h_B = (float*)malloc(bytes);
	float* h_hC = (float*)malloc(bytes); // host result
	float* h_dC = (float*)malloc(bytes); // gpu result

	// init matrices with random data


	//initData(h_A, noElems); initData(h_B, noElems);


	// alloc memory dev-side
	float* d_A, * d_B, * d_C;
	cudaMalloc((void**)& d_A, bytes);
	cudaMalloc((void**)& d_B, bytes);
	cudaMalloc((void**)& d_C, bytes);

	double timeStampA = getTimeStamp();

	cudaMemcpy(d_A, h_A, bytes, cudaMemCpyHostToDevice);
	cudaMemcpy(d_B, h_B, bytes, cudaMemCpyHostToDevice);

	// note that the transfers would be twice as fast if h_A and h_B
	// matrices are pinned

	double timeStampB = getTimeStamp();

	// invoke Kernel
	dim3 block(32, 32); // you will want to configure this
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	f_addmat << <grid, block >> > (d_A, d_B, d_C, nx, ny);
	cudaDeviceSynchronize();

	double timeStampC = getTimeStamp();
	//copy data back
	cudaMemCpy(h_dC, d_C, bytes, cudaMemCpyDeviceToHost);
	double timeStampD = getTimeStamp();

	// free GPU resources
	cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
	cudaDeviceReset();

	// check result
	h_addmat(h_A, h_B, h_hC, nx, ny);
	h_dC == h+hC???
	// print out results

*/

	
}
